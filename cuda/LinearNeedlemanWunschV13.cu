#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"
#include "../c++/timing.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32
#define BATCH_SIZE 500

__global__ void 
needleman_wunsch_kernel(
    int *similarityScores,
    int *stringSpacing,
    directionMain *batchBacktrackMatrices,
    int *batchIndices,
    char *backtrackStringsRet, 
    const char *allSequences, const seqPair *allSequenceInfo,
    const int matchWeight, const int mismatchWeight, const int gapWeight,
    const int startingSequenceIdx, const int stringLengthMax)
{

    const int tid = threadIdx.x;

    extern __shared__ int warpEdgeScore[]; 

    // We are launching multiple blocks, each of a warp of threads
    // Each block handles their own sequence alignment
    // We index into the array to obtain the strings and length
    
    int batchIndex = batchIndices[blockIdx.x];
    directionMain *backtrackMatrix = batchBacktrackMatrices + batchIndex;

    const int sequenceIdx = startingSequenceIdx + blockIdx.x;
    const seqPair sequenceInfo = allSequenceInfo[sequenceIdx];
    
    const char *queryString = allSequences + sequenceInfo.queryIdx;
    const char *referenceString = allSequences + sequenceInfo.referenceIdx;

    // The matrices are of size (queryLength + 1) * (referenceLength + 1)
    const int numRows = sequenceInfo.querySize + 1;
    const int numCols = sequenceInfo.referenceSize + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols) {
        backtrackMatrix[elementIdx] = QUERY_INSERTION;
        elementIdx += BLOCK_SIZE;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        backtrackMatrix[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += BLOCK_SIZE;
    }

    if (tid == 0) {
        backtrackMatrix[0] = NONE_MAIN;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    int leftDiag = gapWeight*tid;
    int left = gapWeight*(tid+1);
    int up = gapWeight*(tid+1); 

    // Going through all of the rows each thread has to do
    for (int stripeStart = 1; stripeStart < numRows; stripeStart+=BLOCK_SIZE){

        int row = stripeStart + tid;
        int largestScore;

        /* threads outside of bounds should abort */
        if (row >= numRows) return;

        leftDiag = gapWeight*(row - 1);
        left = gapWeight*(row);

        for (int col = 1; col < (numCols+numRows); ++col){
            
            int adj_col = col - tid;

            if (row == 1){
                leftDiag = gapWeight*(adj_col - 1);
                up = gapWeight*(adj_col);
            }

            /* for all but the first stripe, t0 must grab its diagonal and upper values from t31 */
            if (stripeStart > 1 && tid == 0 && adj_col < numCols){
                up = warpEdgeScore[adj_col];
                leftDiag = (adj_col == 1) ? gapWeight*(row - 1) : warpEdgeScore[adj_col - 1];
            }

            if (adj_col > 0 && adj_col < numCols){
                largestScore = 0;
                char queryChar = queryString[row-1];
                char referenceChar = referenceString[adj_col-1];
    
                directionMain cornerDirection = NONE_MAIN;
                bool pred;
                bool isMatch = (queryChar == referenceChar);
                cornerDirection = isMatch ? MATCH : MISMATCH;
    
                int matchMismatchScore = isMatch ? leftDiag + matchWeight : leftDiag + mismatchWeight;
                int queryDeletionScore = up + gapWeight;
                int queryInsertionScore = left + gapWeight;
    
                largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
                if (pred) cornerDirection = QUERY_DELETION;
                        
                largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
                if (pred) cornerDirection = QUERY_INSERTION;

                // scoringMatrix[row * numCols + adj_col] = largestScore;
                backtrackMatrix[row * numCols + adj_col] = cornerDirection;

                left = largestScore;

                /* last thread in warp stores its scores in shared memory for t0 to access */
                if (tid == 31){
                    warpEdgeScore[adj_col] = largestScore;
                }

                leftDiag = up;
            }

            /*  top value for thread n + 1 is thread n's largestScore (just calculated value)*/
            up = __shfl_up_sync(0xffffffff, largestScore, 1);
        }

        if (row == numRows-1) {
            similarityScores[blockIdx.x] = largestScore;
        }
    }

    /* --- (END) POPULATING THE SCORING MATRIX -- */

    /* --- (BEGIN) DETERMINING BACKTRACKING -- */

    // Starting at the end
    if (tid == 0) {

        int referenceStrIdx = (stringLengthMax * 3) * blockIdx.x + (stringLengthMax-1);
        int alignmentStrIdx = referenceStrIdx + stringLengthMax;
        int queryStrIdx = alignmentStrIdx + stringLengthMax;

        backtrackStringsRet[referenceStrIdx] = '\0';
        backtrackStringsRet[alignmentStrIdx] = '\0';
        backtrackStringsRet[queryStrIdx] = '\0';

        int currentMemoRow = numRows - 1;
        int currentMemoCol = numCols - 1;

        while ((currentMemoRow != 0) || (currentMemoCol != 0)) {

            referenceStrIdx--;
            alignmentStrIdx--;
            queryStrIdx--;
            
            // Determine the current cell's predecessor
            switch (backtrackMatrix[(currentMemoRow * numCols) + currentMemoCol]) {
                
                case MATCH:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '*';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if match

                case MISMATCH: 
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '|';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if mismatch
                
                case QUERY_DELETION:
                    backtrackStringsRet[referenceStrIdx] = '_';
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    break;
                // end if query deletion
                
                case QUERY_INSERTION:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = '_';
                    --currentMemoCol;
                    break;
                // end if query insertion
                
                default:
                    printf("ERROR\n");
                    return;
                // end if upper gap

            } // end switch
        } // end while

        stringSpacing[blockIdx.x] = referenceStrIdx;
    }
}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    // printf("[Cuda Details]\n");
    // int deviceCount;
    // hipError_t err = hipGetDeviceCount(&deviceCount);
    // if (err != hipSuccess) {
    //     printf("FAILED TO GET DEVICE COUNT\n");
    //     printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
    //     exit(1);
    // }

    // printf("Device count: %d\n", deviceCount);
    // int device = 0;
    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, device);
    // printf("Device %d has compute capability %d.%d.\n",
    //        device, deviceProp.major, deviceProp.minor);
    // printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapWeight       = -2;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-gap") == 0) {
        gapWeight = atoi(argv[8]);
    }

    // Parse input file
    // printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* allSequenceInfo;
    char* sequences;
    fileInfo = parseInput(pairFileName, allSequenceInfo, sequences);
    // printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    // Start timer
    uint64_t kernel_time = 0;
    uint64_t memalloc_time = 0;
    uint64_t backtracking_time = 0;
    uint64_t start_time = start_timer();
    
    // Copy over the sequences
    char* deviceSequences;
    seqPair *deviceAllSequenceInfo;

    uint64_t start_memalloc = get_time();
    handleErrs(
        hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMalloc(&deviceAllSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceAllSequenceInfo, allSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    /* 
    store all backtracking matrices for a batch in one matrix - each warp will index in via index array
    */

    const int numStreamsConcurrently = 10;

    const int totalNumBatches = fileInfo.numPairs/BATCH_SIZE;
    const int totalNumberStreamBatches = totalNumBatches/numStreamsConcurrently;

    int *deviceBacktrackingIndices;
    int *hostBacktrackingIndices = (int *)malloc(numStreamsConcurrently * BATCH_SIZE * sizeof(int));
    
    handleErrs(
        hipMalloc(&deviceBacktrackingIndices, numStreamsConcurrently * BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceBacktrackingIndices\n"
    );
    
    int *deviceSimilarityScores;
    int *hostSimilarityScores = (int*)malloc(numStreamsConcurrently * BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceSimilarityScores, numStreamsConcurrently * BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );
    
    int *deviceStringSpacing;
    int *hostStringSpacing = (int*)malloc(numStreamsConcurrently * BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceStringSpacing, numStreamsConcurrently * BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );

    // HOLDS ALL OF THE LARGEST REFERENCES AND QUERIES OF EACH BATCH
    int* largestLengthsHolder = (int*)malloc(2 * numStreamsConcurrently * sizeof(int));

    memalloc_time += get_time() - start_memalloc;

    for (size_t streamBatchIdx = 0; streamBatchIdx < totalNumberStreamBatches; ++streamBatchIdx) {

        // Allocating the total backtracking memory for all 10 batches first
        int totalVectorSize = 0;
        
        int largestReferenceLength = 0;
        int largestQueryLength = 0;
        
        int startingSequenceIdx = (streamBatchIdx * numStreamsConcurrently * BATCH_SIZE);

        hostBacktrackingIndices[0] = 0;

        for (size_t streamIdx = 0; streamIdx < numStreamsConcurrently; ++streamIdx) {
        
            int largestReferenceLength = 0;
            int largestQueryLength = 0;

            const int sequenceIdx = startingSequenceIdx + (streamIdx * BATCH_SIZE);

            for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
                const int queryLength = allSequenceInfo[i].querySize;
                const int referenceLength = allSequenceInfo[i].referenceSize;

                largestReferenceLength = max(largestReferenceLength, referenceLength);
                largestQueryLength = max(largestQueryLength, queryLength);

                /* make sure we don't go over the end of the array */
                batchMatrixSize += ((referenceLength + 1) * (queryLength + 1));
                if ((i - startingSequenceIdx) < (numStreamsConcurrently * BATCH_SIZE - 1)){
                    hostBacktrackingIndices[i-startingSequenceIdx + 1] = batchMatrixSize;
                }
            }

            largestLengthsHolder[i*2] = largestReferenceLength;
            largestLengthsHolder[i*2 + 1] = largestQueryLength;
        }


        // For each stream batch, we will handle 10 kernels at a time (10 streams running concurrently)
        hipStream_t streams[numStreamsConcurrently];
        
        for (int streamIdx = 0; streamIdx < numStreamsConcurrently; ++streamIdx) {
            hipStreamCreate(&streams[streamIdx]);
        }

        // Performfing the async calls
        for (int streamIdx = 0; streamIdx < numStreamsConcurrently; ++streamIdx) {

            // 1. Stream copies over their own section of backtracking indices
            handleErrs(
                hipMemcpyAsync(deviceBacktrackingIndices, hostBacktrackingIndices, BATCH_SIZE * sizeof(int), 
                                hipMemcpyHostToDevice, streams[streamIdx]),
                "FAILED TO COPY MEMORY FOR deviceBacktrackingIndices\n"
            );

            handleErrs(
                hipMemsetAsync(deviceBacktrackStringRet, 0, (stringLengthMax * 3) * BATCH_SIZE * sizeof(char), 
                                streams[streamIdx]),
                "FAILED TO memset deviceMatricesAll\n"
            );

            int smem_size = (largestReferenceLength + 1) * sizeof(int);
            needleman_wunsch_kernel<<<BATCH_SIZE, BLOCK_SIZE, smem_size>>>(
                deviceSimilarityScores,
                deviceStringSpacing,
                deviceMatricesAll,
                deviceBacktrackingIndices,
                deviceBacktrackStringRet,
                deviceSequences, deviceAllSequenceInfo,
                matchWeight, mismatchWeight, gapWeight,
                sequenceIdx, stringLengthMax
            );

            handleErrs(
                hipMemcpyAsync(hostSimilarityScores, deviceSimilarityScores, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost, 
                                streams[streamIdx]),
                "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
            );

            handleErrs(
                hipMemcpyAsync(hostStringSpacing, deviceStringSpacing, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost, 
                                streams[streamIdx]),
                "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
            );

            char *hostBacktrackingStringRet = (char *)malloc(stringLengthMax * 3 * BATCH_SIZE * sizeof(char));

            handleErrs(
                hipMemcpyAsync(hostBacktrackingStringRet, deviceBacktrackStringRet, (stringLengthMax * 3) * BATCH_SIZE * sizeof(char),
                                hipMemcpyDeviceToHost, streams[streamIdx]),
                "FAILED TO COPY BACKTRACKING STRING FROM DEVICE --> HOST\n"
            );
        }

        for (int streamIdx = 0; streamIdx < numStreamsConcurrently; ++streamIdx) {

            hipStreamSynchronize(streams[streamIdx]);

            const int sequenceIdx = startingSequenceIdx + (streamIdx * BATCH_SIZE);

            for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
                // Backtrack matrices
                printf("%d | %d\n", i, hostSimilarityScores[i-sequenceIdx]);

                int spacing = hostStringSpacing[i-sequenceIdx];

                printf("%s\n", hostBacktrackingStringRet + spacing);
                printf("%s\n", hostBacktrackingStringRet + stringLengthMax + spacing);
                printf("%s\n", hostBacktrackingStringRet + stringLengthMax + stringLengthMax + spacing);
            }

            hipStreamDestroy(streams[streamIdx]);
        }
    }

    free(largestLengthsHolder);

    free(hostBacktrackingStringRet);
    hipFree(deviceBacktrackStringRet);
    hipFree(deviceMatricesAll);

    free(hostBacktrackingIndices);
    free(hostSimilarityScores);
    free(hostStringSpacing);

    hipFree(deviceBacktrackingIndices);
    hipFree(deviceSimilarityScores);
    hipFree(deviceStringSpacing);

    hipFree(deviceSequences);
    hipFree(deviceAllSequenceInfo);

    uint64_t elapsed_time = get_elapsed_time();
    printf("Elapsed time (usec): %lld\n", elapsed_time);
    printf("Elapsed kernel time (usec): %lld\n", kernel_time);
    printf("Elapsed backtracking time (usec): %lld\n", backtracking_time);
    printf("Elapsed memallocing time (usec): %lld\n", memalloc_time);
    printf("Elapsed time sum (usec): %lld\n",kernel_time + backtracking_time + memalloc_time);

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(allSequenceInfo, sequences);
}