#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32

// Defing this will test all of the sequences in the input file
#define TEST_ALL

__global__ void 
needleman_wunsch_kernel_warp_shuffle(
    int * scoringMatrix,
    direction * __restrict__ backtrackMatrix,
    const char * __restrict__ queryString, const char * __restrict__ referenceString,
    const int queryLength, const int referenceLength,
    const int matchWeight, const int mismatchWeight, 
    const int gapWeight)
{
    // We are only launching 1 block
    // Thus, each thread will only have a unique threadID that differentiates the threads

    const int tid = threadIdx.x;
    const int threadCount = blockDim.x;

    const int numRows = queryLength + 1;
    const int numCols = referenceLength + 1;

    extern __shared__ int warpEdgeScore[];

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */
    // Used for when a thread has to iterate over more than one col/row
    int elementIdx = tid;

    // Initialize the top row
    // Writing in DRAM burst for faster updating

    while(elementIdx < numCols) {
        scoringMatrix[elementIdx] = gapWeight*elementIdx;
        backtrackMatrix[elementIdx] = QUERY_INSERTION;
        elementIdx += threadCount;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        scoringMatrix[elementIdx*numCols] = gapWeight*elementIdx;
        backtrackMatrix[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += threadCount;
    }

    if (tid == 0) {
        backtrackMatrix[0] = NONE;
    }

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */
    int leftDiag = gapWeight*tid, left = gapWeight*(tid+1), up = gapWeight*(tid+1), tmp_left = left;

    for (int stripeStart = 1; stripeStart < numRows; stripeStart+=threadCount){

        int row = stripeStart + tid;

        /* threads outside of bounds should abort */
        if (row >= numRows) return;

        for (int col = 1; col < (numCols+numRows); ++col){
            int largestScore = 0;
            int adj_col = col - tid;

            if (row == 1){
                leftDiag = gapWeight*(adj_col - 1);
                up = gapWeight*(adj_col);
            }

            if (adj_col == 1){
                leftDiag = gapWeight*(row - 1);
                left = gapWeight*(row);
            }

            /* for all but the first stripe, t0 must grab its diagonal and upper values from t31 */
            if (stripeStart > 1 && tid == 0 && adj_col < numCols){
                up = warpEdgeScore[adj_col];
                leftDiag = (adj_col == 1) ? gapWeight*(row - 1) : warpEdgeScore[adj_col - 1];
            }

            if (adj_col > 0 && adj_col < numCols){
                char queryChar = queryString[row-1];
                char referenceChar = referenceString[adj_col-1];
    
                direction cornerDirection = NONE;
                bool pred;
                bool isMatch = (queryChar == referenceChar);
                cornerDirection = isMatch ? MATCH : MISMATCH;
    
                int matchMismatchScore = isMatch ? leftDiag + matchWeight : leftDiag + mismatchWeight;
                int queryDeletionScore = up + gapWeight;
                int queryInsertionScore = left + gapWeight;
    
                largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
                if (pred) cornerDirection = QUERY_DELETION;
                        
                largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
                if (pred) cornerDirection = QUERY_INSERTION;

                scoringMatrix[row * numCols + adj_col] = largestScore;
                backtrackMatrix[row * numCols + adj_col] = cornerDirection;

                tmp_left = left;
                left = largestScore;

                /* last thread in warp stores its scores in shared memory for t0 to access */
                if (tid == 31){
                    warpEdgeScore[adj_col] = largestScore;
                }
            }

            /*  top value for thread n + 1 is thread n's largestScore (just calculated value)*/
            up = __shfl_up_sync(0xffffffff, largestScore, 1);

            /* left diag value for thread n + 1 is thread n's left value (previously calculated value) */
            leftDiag = __shfl_up_sync(0xffffffff, tmp_left, 1);
        }
    }
}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapWeight       = -2;
    int threadCount     = 32;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-gap") == 0) {
        gapWeight = atoi(argv[8]);
    }
    if(argc > 9 && strcmp(argv[9], "-threads-per-alignment") == 0) {
        threadCount = atoi(argv[10]);
    }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* sequenceIdxs;
    char* sequences;
    fileInfo = parseInput(pairFileName, sequenceIdxs, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    #ifdef TEST_ALL
        
        // Copy over the sequences
        char* deviceSequences;

        handleErrs(
            hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
        );

        handleErrs(
            hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
        );

        // Run the kernel on every sequence
        for(size_t i = 0; i < fileInfo.numPairs; i++){

            char *referenceString = &sequences[sequenceIdxs[i].referenceIdx];
            char *queryString = &sequences[sequenceIdxs[i].queryIdx];

            int referenceLength = strlen(referenceString);
            int queryLength = strlen(queryString);

            int *deviceScoringMatrix;
            direction *deviceBacktrackMatrix;

            handleErrs(
                hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
                "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
            );
    
            handleErrs(
                hipMalloc(&deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(direction)),
                "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
            );

            // Need to launch kernel
            /* allocate enough shared memory to store 1 row of scores */
            int smem_size = (referenceLength+1)*sizeof(int);
            needleman_wunsch_kernel_warp_shuffle<<<1, BLOCK_SIZE, smem_size>>>(
                deviceScoringMatrix, deviceBacktrackMatrix,
                deviceSequences + sequenceIdxs[i].queryIdx, deviceSequences + sequenceIdxs[i].referenceIdx, 
                sequenceIdxs[i].querySize, sequenceIdxs[i].referenceSize, 
                matchWeight, mismatchWeight, gapWeight
            );
            
            // Wait for kernel to finish
            handleErrs(
                hipDeviceSynchronize(),
                "SYNCHRONIZATION FAILED\n"
            );

            // Copy the matrices back over
            int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
            direction *hostBacktrackMatrix = new direction[(referenceLength+1) * (queryLength+1)];

            // Copy information back from device --> host
            handleErrs(
                hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
                "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST"
            );
            
            handleErrs(
                hipMemcpy(hostBacktrackMatrix, deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(direction), hipMemcpyDeviceToHost),
                "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST"
            );

            hipFree(deviceScoringMatrix);
            hipFree(deviceBacktrackMatrix);

            // Backtrack matrices
            printf("%d | %d\n", i, hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
            backtrackNW(hostBacktrackMatrix, referenceString, referenceLength, queryString, queryLength);

            // Free data arrays
            delete[] hostScoringMatrix;
            delete[] hostBacktrackMatrix;
        }

        hipFree(deviceSequences);
    #else
        char *referenceString = &sequences[sequenceIdxs[0].referenceIdx];
        char *queryString = &sequences[sequenceIdxs[0].queryIdx];
        // char *referenceString = "GTCATGCAATAACG";
        // char *queryString = "ATGCAATA";
        // char *referenceString = "GTCAGTA";
        // char *queryString = "ATACA";

        int referenceLength = strlen(referenceString);
        int queryLength = strlen(queryString);

        printf("Reference String: %s (Length: %d)\n", referenceString, referenceLength);
        printf("Query String: %s (Length: %d)\n", queryString, queryLength);
        printf("(MATCH WEIGHT, MISMATCH WEIGHT, GAP WEIGHT): (%d, %d, %d)\n\n", matchWeight, mismatchWeight, gapWeight);

        // Allocate device memory for matrices
        printf("[Allocating CUDA Memory]\n");
        int *deviceScoringMatrix;
        direction *deviceBacktrackMatrix;
        char *deviceReferenceString;
        char *deviceQueryString;

        handleErrs(
            hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(direction)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceReferenceString, (referenceLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceReferenceString, referenceString, (referenceLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryString, (queryLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO QUERY STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceQueryString, queryString, (queryLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO QUERY STRING\n"
        );

        // Need to launch sinular kernel
        // Launching a kernel with 1 block with threadCount threads to populate scoring matrix
        int smem_size = (referenceLength+1)*sizeof(int);
        needleman_wunsch_kernel_warp_shuffle<<<1, BLOCK_SIZE, smem_size>>>(
            deviceScoringMatrix, deviceBacktrackMatrix,
            deviceSequences + sequenceIdxs[i].queryIdx, deviceSequences + sequenceIdxs[i].referenceIdx, 
            sequenceIdxs[i].querySize, sequenceIdxs[i].referenceSize, 
            matchWeight, mismatchWeight, gapWeight
        );

        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );

        // Allocate host memory for matrices
        // Allow for matrices to come from device -> host
        // Free up device memory
        int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
        direction *hostBacktrackMatrix = new direction[(referenceLength+1) * (queryLength+1)];

        // Copy information back from device --> host
        handleErrs(
            hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST"
        );
        
        handleErrs(
            hipMemcpy(hostBacktrackMatrix, deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(direction), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST"
        );

        hipFree(deviceScoringMatrix);
        hipFree(deviceBacktrackMatrix);
        hipFree(deviceQueryString);
        hipFree(deviceReferenceString);

        // Print Matrix
        printf("Scored Matrix\n");
        printMatrix(hostScoringMatrix, referenceLength + 1, queryLength + 1);
        printf("Backtrack Matrix\n");
        printBacktrackMatrix(hostBacktrackMatrix, referenceLength + 1, queryLength + 1);
        

        // Perform backtracking on host and print results
        printf("0 | %d\n", hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
        backtrackNW(hostBacktrackMatrix, referenceString, referenceLength, queryString, queryLength);
        
        // Free data arrays
        delete[] hostScoringMatrix;
        delete[] hostBacktrackMatrix;
    #endif

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(sequenceIdxs, sequences);
}