#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"
#include "../c++/timing.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32

// Defining this will test all of the sequences in the input file
#define TEST_ALL

/*
    THINGS TO CONSIDER FOR OPTIMIZATION
    1. Complete removal of the scoring matrix altogether (Use of warp shuffling and shared memory)
    2. Using 16x2 DPX instructions to have a thread work on 2 cells concurrently

*/

// NEEDLEMAN WUNSCH BASELINE KERNEL

__global__ void 
affine_needleman_wunsch_kernel(
    int *scoringMatrix, directionMain *backtrackScoringMatrix,
    int *queryDeletionMatrix, directionIndel *backtrackDeletionMatrix,
    int *queryInsertionMatrix, directionIndel *backtrackInsertionMatrix,
    const char *queryString, const char *referenceString,
    const int queryLength, const int referenceLength,
    const int matchWeight, const int mismatchWeight, 
    const int gapOpenWeight, const int gapExtendWeight)
{
    // We are only launching 1 block
    // Thus, each thread will only have a unique threadID that differentiates the threads
    const int tid = threadIdx.x;
    const int threadCount = blockDim.x;

    // The matrices are of size (queryLength + 1) * (referenceLength + 1)
    const int numRows = queryLength + 1;
    const int numCols = referenceLength + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols) {
        scoringMatrix[elementIdx] = gapOpenWeight + (elementIdx * gapExtendWeight);
        backtrackScoringMatrix[elementIdx] = QUERY_INSERTION;
        elementIdx += threadCount;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        scoringMatrix[elementIdx*numCols] = gapOpenWeight + (elementIdx * gapExtendWeight);
        backtrackScoringMatrix[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += threadCount;
    }

    if (tid == 0) {
        backtrackScoringMatrix[0] = NONE_MAIN;
        backtrackDeletionMatrix[0] = NONE_INDEL;
        backtrackInsertionMatrix[0] = NONE_INDEL;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    // Each thread needs to iterate through the loop to be able to make the __syncthreads() call
    // All threads need to be able to reach the __syncthreads() call
    const int differentRows = ((numRows - 1) / BLOCK_SIZE) + 1;

    // Every thread gets a row and char
    int rowIdx = tid + 1;
    char queryChar = queryString[rowIdx - 1];

    int cell00Idx;
    int cell01Idx;
    int cell10Idx;
    int cell11Idx;

    for (int rowLoopIdx = 0; rowLoopIdx < differentRows; ++rowLoopIdx) {

        // If the thread in the warp is outside the matrix, wait for the other threads
        if (rowIdx < numRows) {

            // Each later thread must wait for the previous thread
            int adjCol = 1 - tid;

            // Setup cell indices once a thread can start executing
            cell00Idx = (rowIdx-1)*numCols;
            cell01Idx = (rowIdx-1)*numCols + 1;
            cell10Idx = rowIdx*numCols;
            cell11Idx = rowIdx*numCols + 1;
            
            // Each thread must go through the whole row
            // BUT, there is an adjustment that each thread must wait for
            for (int colIdx = 1; colIdx < (numCols+BLOCK_SIZE); ++colIdx) {

                // Main cell updating
                if((adjCol > 0) && (adjCol < numCols)){

                    bool pred;

                    // Handling scores of performing an query deletion at the end
                    // Calculating best score of either creating or extending the deletion gap
                    if (rowIdx == 1) {
                        // PROBABLY CAN HANDLE ROW 1 DURING INITIALIZATION PHASE
                        // Always assuming just opening new gap
                        queryDeletionMatrix[cell11Idx] = scoringMatrix[cell01Idx] + gapOpenWeight + gapExtendWeight;
                        backtrackDeletionMatrix[cell11Idx] = GAP_OPEN;
                    } else {
                        queryDeletionMatrix[cell11Idx] = __vibmax_s32(
                            scoringMatrix[cell01Idx] + gapOpenWeight + gapExtendWeight,  // Opening new gap at the end
                            queryDeletionMatrix[cell01Idx] + gapExtendWeight,            // Extending current gap at end
                            &pred
                        );
                        backtrackDeletionMatrix[cell11Idx] = pred ? GAP_OPEN : GAP_EXTEND;
                    }

                    // Handling scores of performing an query insertion at the end
                    // Calculating best score of either creating or extending the insertion gap
                    if (colIdx == 1) {
                        // PROBABLY CAN HANDLE COL 1 DURING INITIALIZATION PHASE
                        // Always assuming just opening new gap
                        queryInsertionMatrix[cell11Idx] = scoringMatrix[cell10Idx] + gapOpenWeight + gapExtendWeight;
                        backtrackInsertionMatrix[cell11Idx] = GAP_OPEN;
                    } else {
                        queryInsertionMatrix[cell11Idx] = __vibmax_s32(
                            scoringMatrix[cell10Idx] + gapOpenWeight + gapExtendWeight,  // Opening new gap at the end
                            queryInsertionMatrix[cell10Idx] + gapExtendWeight,           // Extending current gap at end
                            &pred
                        );
                        backtrackInsertionMatrix[cell11Idx] = pred ? GAP_OPEN : GAP_EXTEND;
                    }
                    
                    char referenceChar = referenceString[adjCol - 1];
                    directionMain cornerDirection;

                    // Determine if match
                    bool isMatch = (queryChar == referenceChar);
                    cornerDirection = isMatch ? MATCH : MISMATCH;
    
                    // Get all the possible scores
                    int matchMismatchScore = isMatch ? scoringMatrix[cell00Idx] + matchWeight : scoringMatrix[cell00Idx] + mismatchWeight;
            
                    int largestScore = __vibmax_s32(queryDeletionMatrix[cell11Idx], matchMismatchScore, &pred);
                    if (pred) cornerDirection = QUERY_DELETION;
                    
                    largestScore = __vibmax_s32(queryInsertionMatrix[cell11Idx], largestScore, &pred);
                    if (pred) cornerDirection = QUERY_INSERTION;

                    scoringMatrix[cell11Idx] = largestScore;
                    backtrackScoringMatrix[cell11Idx] = cornerDirection;
                    
                    cell00Idx += 1;
                    cell01Idx += 1;
                    cell10Idx += 1;
                    cell11Idx += 1;
                }

                ++adjCol;

            } // end

        } // end if

        // All previous threads must finish before moving onto the next row
        __syncthreads();

        rowIdx += BLOCK_SIZE;
        queryChar = queryString[rowIdx - 1];

    } // end for

    /* --- (END) POPULATING THE SCORING MATRIX -- */
}


void
handleErrs(
    hipError_t err,
    const char *errMsg)
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapOpenWeight   = -3;
    int gapExtendWeight = -1;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-open") == 0) {
        gapOpenWeight = atoi(argv[8]);
    }
    if(argc > 9 && strcmp(argv[9], "-extend") == 0) {
        gapExtendWeight = atoi(argv[10]);
    }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* sequenceIdxs;
    char* sequences;
    fileInfo = parseInput(pairFileName, sequenceIdxs, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    // Start timer
    uint64_t kernel_time = 0;
    uint64_t memalloc_time = 0;
    uint64_t backtracking_time = 0;
    uint64_t start_time = start_timer();
        
    // Copy over the sequences
    char* deviceSequences;
    uint64_t start_memalloc = get_time();
    handleErrs(
        hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    memalloc_time += get_time() - start_memalloc;

    // Run the kernel on every sequence
    for(size_t i = 0; i < fileInfo.numPairs; i++){
        start_memalloc = get_time();

        char *referenceString = &sequences[sequenceIdxs[i].referenceIdx];
        char *queryString = &sequences[sequenceIdxs[i].queryIdx];

        int referenceLength = strlen(referenceString);
        int queryLength = strlen(queryString);

        int *deviceScoringMatrix;
        directionMain *deviceBacktrackMatrix;

        int *deviceQueryDeletionMatrix;
        directionIndel *deviceDeletionBacktrackMatrix;

        int *deviceQueryInsertionMatrix;
        directionIndel *deviceInsertionBacktrackMatrix;

        handleErrs(
            hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryDeletionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceDeletionBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryInsertionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceInsertionBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        memalloc_time += get_time() - start_memalloc;
        uint64_t start_kernel = get_time();

        // Need to launch kernel
        affine_needleman_wunsch_kernel<<<1, BLOCK_SIZE>>>(
            deviceScoringMatrix, deviceBacktrackMatrix,
            deviceQueryDeletionMatrix, deviceDeletionBacktrackMatrix,
            deviceQueryInsertionMatrix, deviceInsertionBacktrackMatrix,
            deviceSequences + sequenceIdxs[i].queryIdx, deviceSequences + sequenceIdxs[i].referenceIdx, 
            sequenceIdxs[i].querySize, sequenceIdxs[i].referenceSize, 
            matchWeight, mismatchWeight, gapOpenWeight, gapExtendWeight
        );
        
        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );

        kernel_time += get_time() - start_kernel;
        start_memalloc = get_time();

        // Copy the matrices back over
        int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
        directionMain *hostBacktrackMatrix = new directionMain[(referenceLength+1) * (queryLength+1)];

        int *hostQueryDeletionMatrix = new int[(referenceLength+1) * (queryLength+1)];
        directionIndel *hostDeletionBacktrackMatrix = new directionIndel[(referenceLength+1) * (queryLength+1)];

        int *hostQueryInsertionMatrix = new int[(referenceLength+1) * (queryLength+1)];
        directionIndel *hostInsertionBacktrackMatrix = new directionIndel[(referenceLength+1) * (queryLength+1)];

        // Copy information back from device --> host
        handleErrs(
            hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
        );
        
        handleErrs(
            hipMemcpy(hostBacktrackMatrix, deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
        );

        hipFree(deviceScoringMatrix);
        hipFree(deviceBacktrackMatrix);

        hipFree(deviceQueryDeletionMatrix);
        hipFree(deviceDeletionBacktrackMatrix);

        hipFree(deviceQueryInsertionMatrix);
        hipFree(deviceInsertionBacktrackMatrix);

        memalloc_time += get_time() - start_memalloc;

        uint64_t start_backtrack = get_time();
        // Backtrack matrices
        printf("%d | %d\n", i, hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
        backtrackANW(hostBacktrackMatrix, hostInsertionBacktrackMatrix, hostDeletionBacktrackMatrix, referenceString, referenceLength, queryString, queryLength);
        backtracking_time += get_time() - start_backtrack;

        // Free data arrays
        delete[] hostScoringMatrix;
        delete[] hostBacktrackMatrix;

        delete[] deviceQueryDeletionMatrix;
        delete[] deviceDeletionBacktrackMatrix;

        delete[] deviceQueryInsertionMatrix;
        delete[] deviceInsertionBacktrackMatrix;
    }

    hipFree(deviceSequences);

    uint64_t elapsed_time = get_elapsed_time();
    printf("Elapsed time (usec): %lld\n", elapsed_time);
    printf("Elapsed kernel time (usec): %lld\n", kernel_time);
    printf("Elapsed backtracking time (usec): %lld\n", backtracking_time);
    printf("Elapsed memallocing time (usec): %lld\n", memalloc_time);
    printf("Elapsed time sum (usec): %lld\n",kernel_time + backtracking_time + memalloc_time);

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(sequenceIdxs, sequences);
}