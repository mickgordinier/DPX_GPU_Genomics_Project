#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32

// Defing this will test all of the sequences in the input file
#define TEST_ALL

/*
    THINGS TO CONSIDER FOR OPTIMIZATION
    1. Complete removal of the scoring matrix altogether (Use of warp shuffling and shared memory)
    2. Using 16x2 DPX instructions to have a thread work on 2 cells concurrently

*/

// NEEDLEMAN WUNSCH BASELINE KERNEL

__global__ void 
affine_needleman_wunsch_kernel(
    int *scoringMatrix, directionMain *scoringBacktrack,
    int *queryInsertionMatrix, directionIndel *queryInsertionBacktrack,
    int *queryDeletionMatrix, directionIndel *queryDeletionBacktrack,
    const char *queryString, const char *referenceString,
    const int queryLength, const int referenceLength,
    const int matchWeight, const int mismatchWeight, 
    const int openWeight, const int extendWeight)
{
    // We are only launching 1 block
    // Thus, each thread will only have a unique threadID that differentiates the threads
    const int tid = threadIdx.x;
    const int threadCount = blockDim.x;

    // The matrices are of size (queryLength + 1) * (referenceLength + 1)
    const int numRows = queryLength + 1;
    const int numCols = referenceLength + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols) {
        scoringMatrix[elementIdx] = openWeight + (extendWeight * elementIdx);
        scoringBacktrack[elementIdx] = QUERY_INSERTION;
        elementIdx += threadCount;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        scoringMatrix[elementIdx*numCols] = openWeight + (extendWeight * elementIdx);
        scoringBacktrack[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += threadCount;
    }

    if (tid == 0) {
        scoringMatrix[0] = 0;
        scoringBacktrack[0] = NONE_MAIN;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    // Each thread needs to iterate through the loop to be able to make the __syncthreads() call
    // All threads need to be able to reach the __syncthreads() call
    const int differentRows = ((numRows - 1) / BLOCK_SIZE) + 1;

    // Every thread gets a row and char
    int rowIdx = tid + 1;
    char queryChar;

    int cell00Idx;
    int cell01Idx;
    int cell10Idx;
    int cell11Idx;

    for (int rowLoopIdx = 0; rowLoopIdx < differentRows; ++rowLoopIdx) {

        // If the thread in the warp is outside the matrix, wait for the other threads
        if (rowIdx < numRows) {

            queryChar = queryString[rowIdx - 1];

            // Each later thread must wait for the previous thread
            int adjCol = 1 - tid;
            
            // Each thread must go through the whole row
            // BUT, there is an adjustment that each thread must wait for
            for (int colIdx = 1; colIdx < (numCols+numRows); ++colIdx) {

                // Setup cell indices once a thread can start executing
                if(adjCol == 1){
                    cell00Idx = (rowIdx-1)*numCols + adjCol - 1;
                    cell01Idx = (rowIdx-1)*numCols + adjCol;
                    cell10Idx = rowIdx*numCols + adjCol - 1;
                    cell11Idx = rowIdx*numCols + adjCol; 
                } 

                // Main cell updating
                if((adjCol > 0) && (adjCol < numCols)){
                    
                    char referenceChar = referenceString[adjCol - 1];
                    directionMain cornerDirection = NONE_MAIN;
                    bool pred;

                    // Handling scores of performing an query deletion at the end
                    // Calculating best score of either creating or extending the deletion gap
                    if (rowIdx == 1) {
                        // PROBABLY CAN HANDLE ROW 1 DURING INITIALIZATION PHASE
                        // Always assuming just opening new gap
                        queryDeletionMatrix[cell11Idx] = scoringMatrix[cell01Idx] + openWeight + extendWeight;
                        queryDeletionBacktrack[cell11Idx] = GAP_OPEN;
                    } else {
                        queryDeletionMatrix[cell11Idx] = __vibmax_s32(
                            scoringMatrix[cell01Idx] + openWeight + extendWeight,  // Opening new gap at the end
                            queryDeletionMatrix[cell01Idx] + extendWeight,            // Extending current gap at end
                            &pred
                        );
                        queryDeletionBacktrack[cell11Idx] = pred ? GAP_OPEN : GAP_EXTEND;
                    }

                    // Handling scores of performing an query insertion at the end
                    // Calculating best score of either creating or extending the insertion gap
                    if (adjCol == 1) {
                        // PROBABLY CAN HANDLE COL 1 DURING INITIALIZATION PHASE
                        // Always assuming just opening new gap
                        queryInsertionMatrix[cell11Idx] = scoringMatrix[cell10Idx] + openWeight + extendWeight;
                        queryInsertionBacktrack[cell11Idx] = GAP_OPEN;
                    } else {
                        queryInsertionMatrix[cell11Idx] = __vibmax_s32(
                            scoringMatrix[cell10Idx] + openWeight + extendWeight,  // Opening new gap at the end
                            queryInsertionMatrix[cell10Idx] + extendWeight,           // Extending current gap at end
                            &pred
                        );
                        queryInsertionBacktrack[cell11Idx] = pred ? GAP_OPEN : GAP_EXTEND;
                    }
                    
                    // Determine if match
                    bool isMatch = (queryChar == referenceChar);
                    cornerDirection = isMatch ? MATCH : MISMATCH;
    
                    // Get all the possible scores
                    int matchMismatchScore = isMatch ? scoringMatrix[cell00Idx] + matchWeight : scoringMatrix[cell00Idx] + mismatchWeight;
                    int queryDeletionScore = queryDeletionMatrix[cell11Idx];
                    int queryInsertionScore = queryInsertionMatrix[cell11Idx];
    
                    // Find the largest of the 3 scores
                    // Utilizing DPX instructions for updating
                    // pred = (queryDeletionScore >= matchMismatchScore)
                    int largestScore;
                    largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
                    if (pred) cornerDirection = QUERY_DELETION;
                    
                    largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
                    if (pred) cornerDirection = QUERY_INSERTION;
    
                    // Update scoring matrix and incrementing pointers
                    scoringMatrix[cell11Idx] = largestScore;
                    scoringBacktrack[cell11Idx] = cornerDirection;
                    cell00Idx += 1;
                    cell01Idx += 1;
                    cell10Idx += 1;
                    cell11Idx += 1;
                }

                ++adjCol;

            } // end 

        } // end if

        // All previous threads must finish before moving onto the next row
        __syncthreads();
            
        rowIdx += BLOCK_SIZE;
        queryChar = queryString[rowIdx - 1];

    } // end for

    /* --- (END) POPULATING THE SCORING MATRIX -- */
}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int openWeight       = -3;
    int extendWeight       = -1;
    // int threadCount     = 32;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-open") == 0) {
        openWeight = atoi(argv[8]);
    }
    if(argc > 9 && strcmp(argv[9], "-extend") == 0) {
        extendWeight = atoi(argv[10]);
    }
    // if(argc > 9 && strcmp(argv[9], "-threads-per-alignment") == 0) {
    //     threadCount = atoi(argv[10]);
    // }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* sequenceIdxs;
    char* sequences;
    fileInfo = parseInput(pairFileName, sequenceIdxs, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    #ifdef TEST_ALL
        
        // Copy over the sequences
        char* deviceSequences;

        handleErrs(
            hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
        );

        handleErrs(
            hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
        );

        // Run the kernel on every sequence
        for(size_t i = 0; i < fileInfo.numPairs; i++){

            char *referenceString = &sequences[sequenceIdxs[i].referenceIdx];
            char *queryString = &sequences[sequenceIdxs[i].queryIdx];

            int referenceLength = strlen(referenceString);
            int queryLength = strlen(queryString);

            int *deviceScoringMatrix;
            int *deviceQueryInsertionMatrix;
            int *deviceQueryDeletionMatrix;
            directionMain *deviceScoringBacktrack;
            directionIndel *deviceQueryInsertionBacktrack;
            directionIndel *deviceQueryDeletionBacktrack;

            handleErrs(
                hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
                "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
            );

            handleErrs(
                hipMalloc(&deviceQueryInsertionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
                "FAILED TO ALLOCATE MEMORY TO INSERTION MATRIX\n"
            );

            handleErrs(
                hipMalloc(&deviceQueryDeletionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
                "FAILED TO ALLOCATE MEMORY TO DELETION MATRIX\n"
            );
    
            handleErrs(
                hipMalloc(&deviceScoringBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionMain)),
                "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
            );
    
            handleErrs(
                hipMalloc(&deviceQueryInsertionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
                "FAILED TO ALLOCATE MEMORY TO INSERTION BACKTRACK MATRIX\n"
            );
    
            handleErrs(
                hipMalloc(&deviceQueryDeletionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
                "FAILED TO ALLOCATE MEMORY TO DELETION BACKTRACK MATRIX\n"
            );

            // Need to launch kernel
            affine_needleman_wunsch_kernel<<<1, BLOCK_SIZE>>>(
                deviceScoringMatrix, deviceScoringBacktrack,
                deviceQueryInsertionMatrix, deviceQueryInsertionBacktrack,
                deviceQueryDeletionMatrix, deviceQueryDeletionBacktrack,
                deviceSequences + sequenceIdxs[i].queryIdx, deviceSequences + sequenceIdxs[i].referenceIdx, 
                sequenceIdxs[i].querySize, sequenceIdxs[i].referenceSize, 
                matchWeight, mismatchWeight, openWeight, extendWeight
            );
            
            // Wait for kernel to finish
            handleErrs(
                hipDeviceSynchronize(),
                "SYNCHRONIZATION FAILED\n"
            );

            // Copy the matrices back over
            int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
            int *hostQueryInsertionMatrix = new int[(referenceLength+1) * (queryLength+1)];
            int *hostQueryDeletionMatrix = new int[(referenceLength+1) * (queryLength+1)];

            directionMain *hostScoringBacktrack = new directionMain[(referenceLength+1) * (queryLength+1)];
            directionIndel *hostQueryInsertionBacktrack = new directionIndel[(referenceLength+1) * (queryLength+1)];
            directionIndel *hostQueryDeletionBacktrack = new directionIndel[(referenceLength+1) * (queryLength+1)];

            // Copy information back from device --> host
            handleErrs(
                hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
                "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
            );
            handleErrs(
                hipMemcpy(hostQueryInsertionMatrix, deviceQueryInsertionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
                "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
            );
            handleErrs(
                hipMemcpy(hostQueryDeletionMatrix, deviceQueryDeletionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
                "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
            );
            
            handleErrs(
                hipMemcpy(hostScoringBacktrack, deviceScoringBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionMain), hipMemcpyDeviceToHost),
                "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
            );
            handleErrs(
                hipMemcpy(hostQueryInsertionBacktrack, deviceQueryInsertionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel), hipMemcpyDeviceToHost),
                "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
            );
            handleErrs(
                hipMemcpy(hostQueryDeletionBacktrack, deviceQueryDeletionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel), hipMemcpyDeviceToHost),
                "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
            );

            hipFree(deviceScoringMatrix);
            hipFree(deviceQueryInsertionMatrix);
            hipFree(deviceQueryDeletionMatrix);
            hipFree(deviceScoringBacktrack);
            hipFree(deviceQueryInsertionBacktrack);
            hipFree(deviceQueryDeletionBacktrack);

            // Backtrack matrices
            printf("%d | %d\n", i, hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
            backtrackANW(
                hostScoringBacktrack, hostQueryInsertionBacktrack, hostQueryDeletionBacktrack, 
                referenceString, referenceLength, 
                queryString, queryLength
            );

            // Free data arrays
            delete[] hostScoringMatrix;
            delete[] hostQueryInsertionMatrix;
            delete[] hostQueryDeletionMatrix;
            delete[] hostScoringBacktrack;
            delete[] hostQueryInsertionBacktrack;
            delete[] hostQueryDeletionBacktrack;
        }

        hipFree(deviceSequences);
    #else
        char *referenceString = &sequences[sequenceIdxs[0].referenceIdx];
        char *queryString = &sequences[sequenceIdxs[0].queryIdx];
        // char *referenceString = "GTCATGCAATAACG";
        // char *queryString = "ATGCAATA";
        // char *referenceString = "GTCAGTA";
        // char *queryString = "ATACA";

        int referenceLength = strlen(referenceString);
        int queryLength = strlen(queryString);

        printf("Reference String: %s (Length: %d)\n", referenceString, referenceLength);
        printf("Query String: %s (Length: %d)\n", queryString, queryLength);
        printf("(MATCH WEIGHT, MISMATCH WEIGHT, GAP OPEN, GAP EXTEND): (%d, %d, %d, %d)\n\n", matchWeight, mismatchWeight, openWeight, extendWeight);

        // Allocate device memory for matrices
        printf("[Allocating CUDA Memory]\n");
        
        int *deviceScoringMatrix;
        int *deviceQueryInsertionMatrix;
        int *deviceQueryDeletionMatrix;
        directionMain *deviceScoringBacktrack;
        directionIndel *deviceQueryInsertionBacktrack;
        directionIndel *deviceQueryDeletionBacktrack;
        
        char *deviceReferenceString;
        char *deviceQueryString;

        handleErrs(
            hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryInsertionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO INSERTION MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryDeletionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO DELETION MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceScoringBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionMain)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryInsertionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
            "FAILED TO ALLOCATE MEMORY TO INSERTION BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryDeletionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel)),
            "FAILED TO ALLOCATE MEMORY TO DELETION BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceReferenceString, (referenceLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceReferenceString, referenceString, (referenceLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryString, (queryLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO QUERY STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceQueryString, queryString, (queryLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO QUERY STRING\n"
        );

        // Need to launch kernel
        affine_needleman_wunsch_kernel<<<1, BLOCK_SIZE>>>(
            deviceScoringMatrix, deviceScoringBacktrack,
            deviceQueryInsertionMatrix, deviceQueryInsertionBacktrack,
            deviceQueryDeletionMatrix, deviceQueryDeletionBacktrack,
            deviceQueryString, deviceReferenceString, 
            queryLength, referenceLength, 
            matchWeight, mismatchWeight, openWeight, extendWeight
        );
        
        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );

        // Copy the matrices back over
        int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
        int *hostQueryInsertionMatrix = new int[(referenceLength+1) * (queryLength+1)];
        int *hostQueryDeletionMatrix = new int[(referenceLength+1) * (queryLength+1)];

        directionMain *hostScoringBacktrack = new directionMain[(referenceLength+1) * (queryLength+1)];
        directionIndel *hostQueryInsertionBacktrack = new directionIndel[(referenceLength+1) * (queryLength+1)];
        directionIndel *hostQueryDeletionBacktrack = new directionIndel[(referenceLength+1) * (queryLength+1)];

        // Copy information back from device --> host
        handleErrs(
            hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
        );
        handleErrs(
            hipMemcpy(hostQueryInsertionMatrix, deviceQueryInsertionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
        );
        handleErrs(
            hipMemcpy(hostQueryDeletionMatrix, deviceQueryDeletionMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
        );
        
        handleErrs(
            hipMemcpy(hostScoringBacktrack, deviceScoringBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionMain), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
        );
        handleErrs(
            hipMemcpy(hostQueryInsertionBacktrack, deviceQueryInsertionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
        );
        handleErrs(
            hipMemcpy(hostQueryDeletionBacktrack, deviceQueryDeletionBacktrack, (referenceLength+1) * (queryLength+1) * sizeof(directionIndel), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
        );

        hipFree(deviceScoringMatrix);
        hipFree(deviceQueryInsertionMatrix);
        hipFree(deviceQueryDeletionMatrix);
        hipFree(deviceScoringBacktrack);
        hipFree(deviceQueryInsertionBacktrack);
        hipFree(deviceQueryDeletionBacktrack);

        // Print Matrix
        printf("DELETION Matrix\n");
        printMatrix(hostQueryDeletionMatrix, referenceLength + 1, queryLength + 1);
        printf("Scored Matrix\n");
        printMatrix(hostScoringMatrix, referenceLength + 1, queryLength + 1);
        printf("INSERTION Matrix\n");
        printMatrix(hostQueryInsertionMatrix, referenceLength + 1, queryLength + 1);
        printf("Backtrack Matrix\n");
        printBacktrackMatrix(hostScoringBacktrack, referenceLength + 1, queryLength + 1);

        // Backtrack matrices
        printf("%d | %d\n", 0, hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
        backtrackANW(
            hostScoringBacktrack, hostQueryInsertionBacktrack, hostQueryDeletionBacktrack, 
            referenceString, referenceLength, 
            queryString, queryLength
        );

        // Free data arrays
        delete[] hostScoringMatrix;
        delete[] hostQueryInsertionMatrix;
        delete[] hostQueryDeletionMatrix;
        delete[] hostScoringBacktrack;
        delete[] hostQueryInsertionBacktrack;
        delete[] hostQueryDeletionBacktrack;
    #endif

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(sequenceIdxs, sequences);
}