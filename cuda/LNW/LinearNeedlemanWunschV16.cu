#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"
#include "../c++/timing.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32
#define BATCH_SIZE 1000


__global__ void 
needleman_wunsch_kernel(
    unsigned int *similarityScores,
    int *stringSpacing,
    directionMain *batchBacktrackMatrices,
    int *batchBacktrackMatrixStartingIndices,
    char *backtrackStringsRet, 
    const char *allSequences, const seqPair *allSequenceInfo,
    const unsigned int matchWeight, const unsigned int mismatchWeight, const unsigned int gapWeight,
    const int startingSequenceIdx, const int stringLengthMax)
{

    const int tid = threadIdx.x;

    extern __shared__ int warpEdgeScore[]; 

    // We are launching multiple blocks, each of a warp of threads
    // Each block handles their own sequence alignment
    // We index into the array to obtain the strings and length
    
    int startingMatrixIdx1 = batchBacktrackMatrixStartingIndices[(2 * blockIdx.x)];
    int startingMatrixIdx2 = batchBacktrackMatrixStartingIndices[(2 * blockIdx.x) + 1];

    directionMain *backtrackMatrix1 = batchBacktrackMatrices + startingMatrixIdx1;
    directionMain *backtrackMatrix2 = batchBacktrackMatrices + startingMatrixIdx2;

    const int sequenceIdx1 = startingSequenceIdx + (2 * blockIdx.x) ;
    const int sequenceIdx2 = startingSequenceIdx + (2 * blockIdx.x) + 1;

    const seqPair sequenceInfo1 = allSequenceInfo[sequenceIdx1];
    const seqPair sequenceInfo2 = allSequenceInfo[sequenceIdx2];

    const char *queryString1 = allSequences + sequenceInfo1.queryIdx;
    const char *queryString2 = allSequences + sequenceInfo2.queryIdx;

    const char *referenceString1 = allSequences + sequenceInfo1.referenceIdx;
    const char *referenceString2 = allSequences + sequenceInfo2.referenceIdx;

    const int numRows1 = sequenceInfo1.querySize + 1;
    const int numRows2 = sequenceInfo2.querySize + 1;

    const int numCols1 = sequenceInfo1.referenceSize + 1;
    const int numCols2 = sequenceInfo2.referenceSize + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols1) {
        backtrackMatrix1[elementIdx] = QUERY_INSERTION;
        elementIdx += BLOCK_SIZE;
    }
    
    elementIdx = tid;
    while(elementIdx < numCols2) {
        backtrackMatrix2[elementIdx] = QUERY_INSERTION;
        elementIdx += BLOCK_SIZE;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows1) {
        backtrackMatrix1[elementIdx*numCols1] = QUERY_DELETION;
        elementIdx += BLOCK_SIZE;
    }

    elementIdx = tid;
    while(elementIdx < numRows2) {
        backtrackMatrix2[elementIdx*numCols2] = QUERY_DELETION;
        elementIdx += BLOCK_SIZE;
    }

    if (tid == 0) {
        backtrackMatrix1[0] = NONE_MAIN;
        backtrackMatrix2[0] = NONE_MAIN;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    unsigned int leftDiag = gapWeight * tid;
    unsigned int left     = gapWeight * (tid + 1);
    unsigned int up       = gapWeight * (tid + 1); 

    // Going through all of the rows each thread has to do
    for (int stripeStart = 1; stripeStart < numRows; stripeStart+=BLOCK_SIZE){

        int row = stripeStart + tid;
        int largestScore;

        /* threads outside of bounds should abort */
        if (row >= numRows) return;

        leftDiag = gapWeight*(row - 1);
        left = gapWeight*(row);

        for (int col = 1; col < (numCols+numRows); ++col){
            
            int adj_col = col - tid;

            if (row == 1){
                leftDiag = gapWeight*(adj_col - 1);
                up = gapWeight*(adj_col);
            }

            /* for all but the first stripe, t0 must grab its diagonal and upper values from t31 */
            if (stripeStart > 1 && tid == 0 && adj_col < numCols){
                up = warpEdgeScore[adj_col];
                leftDiag = (adj_col == 1) ? gapWeight*(row - 1) : warpEdgeScore[adj_col - 1];
            }

            if (adj_col > 0 && adj_col < numCols){
                largestScore = 0;
                char queryChar = queryString[row-1];
                char referenceChar = referenceString[adj_col-1];
    
                directionMain cornerDirection = NONE_MAIN;
                bool pred;
                bool isMatch = (queryChar == referenceChar);
                cornerDirection = isMatch ? MATCH : MISMATCH;
    
                int matchMismatchScore = isMatch ? leftDiag + matchWeight : leftDiag + mismatchWeight;
                int queryDeletionScore = up + gapWeight;
                int queryInsertionScore = left + gapWeight;
    
                largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
                if (pred) cornerDirection = QUERY_DELETION;
                        
                largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
                if (pred) cornerDirection = QUERY_INSERTION;

                // scoringMatrix[row * numCols + adj_col] = largestScore;
                backtrackMatrix[row * numCols + adj_col] = cornerDirection;

                left = largestScore;

                /* last thread in warp stores its scores in shared memory for t0 to access */
                if (tid == 31){
                    warpEdgeScore[adj_col] = largestScore;
                }

                leftDiag = up;
            }

            /*  top value for thread n + 1 is thread n's largestScore (just calculated value)*/
            up = __shfl_up_sync(0xffffffff, largestScore, 1);
        }

        if (row == numRows-1) {
            similarityScores[blockIdx.x] = largestScore;
        }
    }

    /* --- (END) POPULATING THE SCORING MATRIX -- */

    /* --- (BEGIN) DETERMINING BACKTRACKING -- */

    // Starting at the end
    if (tid == 0) {

        int referenceStrIdx = (stringLengthMax * 3) * blockIdx.x + (stringLengthMax-1);
        int alignmentStrIdx = referenceStrIdx + stringLengthMax;
        int queryStrIdx = alignmentStrIdx + stringLengthMax;

        backtrackStringsRet[referenceStrIdx] = '\0';
        backtrackStringsRet[alignmentStrIdx] = '\0';
        backtrackStringsRet[queryStrIdx] = '\0';

        int currentMemoRow = numRows - 1;
        int currentMemoCol = numCols - 1;

        while ((currentMemoRow != 0) || (currentMemoCol != 0)) {

            referenceStrIdx--;
            alignmentStrIdx--;
            queryStrIdx--;
            
            // Determine the current cell's predecessor
            switch (backtrackMatrix[(currentMemoRow * numCols) + currentMemoCol]) {
                
                case MATCH:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '*';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if match

                case MISMATCH: 
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '|';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if mismatch
                
                case QUERY_DELETION:
                    backtrackStringsRet[referenceStrIdx] = '_';
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    break;
                // end if query deletion
                
                case QUERY_INSERTION:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = '_';
                    --currentMemoCol;
                    break;
                // end if query insertion
                
                default:
                    printf("ERROR\n");
                    return;
                // end if upper gap

            } // end switch
        } // end while

        stringSpacing[blockIdx.x] = referenceStrIdx;
    }

}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapWeight       = -2;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-gap") == 0) {
        gapWeight = atoi(argv[8]);
    }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* allSequenceInfo;
    char* sequences;
    fileInfo = parseInput(pairFileName, allSequenceInfo, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    // Start timer
    uint64_t kernel_time = 0;
    uint64_t memalloc_time = 0;
    uint64_t backtracking_time = 0;
    uint64_t start_time = start_timer();
        
    // Copy over the sequences
    char* deviceSequences;
    seqPair *deviceAllSequenceInfo;

    uint64_t start_memalloc = get_time();
    handleErrs(
        hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMalloc(&deviceAllSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceAllSequenceInfo, allSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    /* 
    store all backtracking matrices for a batch in one matrix - each warp will index in via index array
    */
    int *deviceBacktrackingIndices;
    int *hostBacktrackingIndices = (int *)malloc(BATCH_SIZE * sizeof(int));
    
    handleErrs(
        hipMalloc(&deviceBacktrackingIndices, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceBacktrackingIndices\n"
    );
    
    int *deviceSimilarityScores;
    int *hostSimilarityScores = (int*)malloc(BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceSimilarityScores, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );
    
    int *deviceStringSpacing;
    int *hostStringSpacing = (int*)malloc(BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceStringSpacing, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );

    memalloc_time += get_time() - start_memalloc;

    // Run the kernel on every sequence
    for(size_t sequenceIdx = 0; sequenceIdx < fileInfo.numPairs; sequenceIdx+=BATCH_SIZE){
        start_memalloc = get_time();

        int largestReferenceLength = 0;
        int largestQueryLength = 0;

        /* first warp's starting index is 0 */
        uint64_t batchMatrixSize = 0;
        hostBacktrackingIndices[0] = 0;
        for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
            const int queryLength = allSequenceInfo[i].querySize;
            const int referenceLength = allSequenceInfo[i].referenceSize;

            largestReferenceLength = max(largestReferenceLength, referenceLength);
            largestQueryLength = max(largestQueryLength, queryLength);

            /* make sure we don't go over the end of the array */
            batchMatrixSize += ((referenceLength + 1) * (queryLength + 1));
            if ((i - sequenceIdx) < (BATCH_SIZE - 1)){
                hostBacktrackingIndices[i-sequenceIdx + 1] = batchMatrixSize;
            }
        }

        /* copy backtracking indices to device */
        handleErrs(
            hipMemcpy(deviceBacktrackingIndices, hostBacktrackingIndices, BATCH_SIZE * sizeof(int), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR deviceBacktrackingIndices\n"
        );

        /* allocate device mem for all backtracking matrices */
        directionMain *deviceMatricesAll;
        handleErrs(
            hipMalloc(&deviceMatricesAll, batchMatrixSize*sizeof(directionMain)),
            "FAILED TO ALLOCATE MEMORY TO deviceMatricesAll\n"
        );

        int stringLengthMax = (largestReferenceLength+largestQueryLength+1);

        char *deviceBacktrackStringRet;
    
        handleErrs(
            hipMalloc(&deviceBacktrackStringRet, (stringLengthMax * 3) * BATCH_SIZE * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACKING STRINGS\n"
        );

        memalloc_time += get_time() - start_memalloc;

        uint64_t start_kernel = get_time();
        // Need to launch kernel
        int smem_size = (largestReferenceLength + 1) * sizeof(int);
        needleman_wunsch_kernel<<<BATCH_SIZE, BLOCK_SIZE, smem_size>>>(
            deviceSimilarityScores,
            deviceStringSpacing,
            deviceMatricesAll,
            deviceBacktrackingIndices,
            deviceBacktrackStringRet,
            deviceSequences, deviceAllSequenceInfo,
            matchWeight, mismatchWeight, gapWeight,
            sequenceIdx, stringLengthMax
        );
        
        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );
        kernel_time += get_time() - start_kernel;

        start_memalloc = get_time();

        handleErrs(
            hipMemcpy(hostSimilarityScores, deviceSimilarityScores, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
        );

        handleErrs(
            hipMemcpy(hostStringSpacing, deviceStringSpacing, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
        );

        char *hostBacktrackingStringRet = (char *)malloc(stringLengthMax * 3 * BATCH_SIZE * sizeof(char));

        handleErrs(
            hipMemcpy(hostBacktrackingStringRet, deviceBacktrackStringRet, (stringLengthMax * 3) * BATCH_SIZE * sizeof(char), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACKING STRING FROM DEVICE --> HOST\n"
        );

        memalloc_time += get_time() - start_memalloc;

        for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
        
            // Backtrack matrices
            printf("%d | %d\n", i, hostSimilarityScores[i-sequenceIdx]);

            int spacing = hostStringSpacing[i-sequenceIdx];

            printf("%s\n", hostBacktrackingStringRet + spacing);
            printf("%s\n", hostBacktrackingStringRet + stringLengthMax + spacing);
            printf("%s\n", hostBacktrackingStringRet + stringLengthMax + stringLengthMax + spacing);
        }

        free(hostBacktrackingStringRet);
        hipFree(deviceBacktrackStringRet);
        hipFree(deviceMatricesAll);
    }

    hipFree(deviceSequences);
    hipFree(deviceAllSequenceInfo);

    free(hostBacktrackingIndices);
    free(hostSimilarityScores);
    free(hostStringSpacing);

    hipFree(deviceBacktrackingIndices);
    hipFree(deviceSimilarityScores);
    hipFree(deviceStringSpacing);


    uint64_t elapsed_time = get_elapsed_time();
    printf("Elapsed time (usec): %lld\n", elapsed_time);
    printf("Elapsed kernel time (usec): %lld\n", kernel_time);
    printf("Elapsed backtracking time (usec): %lld\n", backtracking_time);
    printf("Elapsed memallocing time (usec): %lld\n", memalloc_time);
    printf("Elapsed time sum (usec): %lld\n",kernel_time + backtracking_time + memalloc_time);

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(allSequenceInfo, sequences);
}