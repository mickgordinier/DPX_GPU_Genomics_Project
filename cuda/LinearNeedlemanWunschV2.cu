#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32

// Defing this will test all of the sequences in the input file
#define TEST_ALL

/*
    THINGS TO CONSIDER FOR OPTIMIZATION
    1. Complete removal of the scoring matrix altogether (Use of warp shuffling and shared memory)
    2. Using 16x2 DPX instructions to have a thread work on 2 cells concurrently

*/

// NEEDLEMAN WUNSCH BASELINE KERNEL

__global__ void 
needleman_wunsch_kernel(
    int *scoringMatrix, directionMain *backtrackMatrix,
    const char *queryString, const char *referenceString,
    const int queryLength, const int referenceLength,
    const int matchWeight, const int mismatchWeight, 
    const int gapWeight)
{
    // We are only launching 1 block
    // Thus, each thread will only have a unique threadID that differentiates the threads
    const int tid = threadIdx.x;
    const int threadCount = blockDim.x;

    // The matrices are of size (queryLength + 1) * (referenceLength + 1)
    const int numRows = queryLength + 1;
    const int numCols = referenceLength + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols) {
        scoringMatrix[elementIdx] = gapWeight*elementIdx;
        backtrackMatrix[elementIdx] = QUERY_INSERTION;
        elementIdx += threadCount;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        scoringMatrix[elementIdx*numCols] = gapWeight*elementIdx;
        backtrackMatrix[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += threadCount;
    }

    if (tid == 0) {
        backtrackMatrix[0] = NONE_MAIN;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    // Each thread has to do ((numRows-1) / BLOCK_SIZE) different rows
    // During a row pass, each thread will be doing numCols amount of work
    // There is an additional numRows lag amount of waiting for staggering the threads
    const int numIterations = (numCols * ((numRows / BLOCK_SIZE) + 1)) + numRows;

    // Every thread gets a row and char
    int rowIdx = tid + 1;
    char queryChar = queryString[rowIdx - 1];

    // Each later thread must wait for the previous thread
    int adjCol = 1 - tid;

    // Setup cell indices once a thread can start executing
    int cell00Idx = (rowIdx-1)*numCols;
    int cell01Idx = (rowIdx-1)*numCols + 1;
    int cell10Idx = rowIdx*numCols;
    int cell11Idx = rowIdx*numCols + 1; 

    for (int iter = 0; iter < numIterations; ++iter) {
        
        if ((rowIdx < numRows) && (adjCol > 0)) {
            
            char referenceChar = referenceString[adjCol - 1];
            directionMain cornerDirection = NONE_MAIN;
            bool pred;
            
            // Determine if match
            bool isMatch = (queryChar == referenceChar);
            cornerDirection = isMatch ? MATCH : MISMATCH;

            // Get all the possible scores
            int matchMismatchScore = isMatch ? scoringMatrix[cell00Idx] + matchWeight : scoringMatrix[cell00Idx] + mismatchWeight;
            int queryDeletionScore = scoringMatrix[cell01Idx] + gapWeight;
            int queryInsertionScore = scoringMatrix[cell10Idx] + gapWeight;

            // Find the largest of the 3 scores
            // Utilizing DPX instructions for updating
            // pred = (queryDeletionScore >= matchMismatchScore)
            int largestScore;
            largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
            if (pred) cornerDirection = QUERY_DELETION;
            
            largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
            if (pred) cornerDirection = QUERY_INSERTION;

            // Update scoring matrix and incrementing pointers
            scoringMatrix[cell11Idx] = largestScore;
            backtrackMatrix[cell11Idx] = cornerDirection;
            cell00Idx += 1;
            cell01Idx += 1;
            cell10Idx += 1;
            cell11Idx += 1;

        }
        
        if (adjCol == numCols-1) {
            rowIdx += 32;
            queryChar = queryString[rowIdx - 1];
            adjCol = min(0, numCols-33);
            
            cell00Idx = (rowIdx-1)*numCols;
            cell01Idx = (rowIdx-1)*numCols + 1;
            cell10Idx = rowIdx*numCols;
            cell11Idx = rowIdx*numCols + 1; 
        }

        ++adjCol;
    }

    __syncthreads();

    /* --- (END) POPULATING THE SCORING MATRIX -- */
}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapWeight       = -2;
    // int threadCount     = 32;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-gap") == 0) {
        gapWeight = atoi(argv[8]);
    }
    // if(argc > 9 && strcmp(argv[9], "-threads-per-alignment") == 0) {
    //     threadCount = atoi(argv[10]);
    // }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* sequenceIdxs;
    char* sequences;
    fileInfo = parseInput(pairFileName, sequenceIdxs, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    #ifdef TEST_ALL
        
        // Copy over the sequences
        char* deviceSequences;

        handleErrs(
            hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
        );

        handleErrs(
            hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
        );

        // Run the kernel on every sequence
        for(size_t i = 0; i < fileInfo.numPairs; i++){

            char *referenceString = &sequences[sequenceIdxs[i].referenceIdx];
            char *queryString = &sequences[sequenceIdxs[i].queryIdx];

            int referenceLength = strlen(referenceString);
            int queryLength = strlen(queryString);

            int *deviceScoringMatrix;
            directionMain *deviceBacktrackMatrix;

            handleErrs(
                hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
                "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
            );
    
            handleErrs(
                hipMalloc(&deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain)),
                "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
            );

            // Need to launch kernel
            needleman_wunsch_kernel<<<1, BLOCK_SIZE>>>(
                deviceScoringMatrix, deviceBacktrackMatrix,
                deviceSequences + sequenceIdxs[i].queryIdx, deviceSequences + sequenceIdxs[i].referenceIdx, 
                sequenceIdxs[i].querySize, sequenceIdxs[i].referenceSize, 
                matchWeight, mismatchWeight, gapWeight
            );
            
            // Wait for kernel to finish
            handleErrs(
                hipDeviceSynchronize(),
                "SYNCHRONIZATION FAILED\n"
            );

            // Copy the matrices back over
            int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
            directionMain *hostBacktrackMatrix = new directionMain[(referenceLength+1) * (queryLength+1)];

            // Copy information back from device --> host
            handleErrs(
                hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
                "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST\n"
            );
            
            handleErrs(
                hipMemcpy(hostBacktrackMatrix, deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain), hipMemcpyDeviceToHost),
                "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST\n"
            );

            hipFree(deviceScoringMatrix);
            hipFree(deviceBacktrackMatrix);

            // Backtrack matrices
            printf("%d | %d\n", i, hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
            backtrackNW(hostBacktrackMatrix, referenceString, referenceLength, queryString, queryLength);

            // Free data arrays
            delete[] hostScoringMatrix;
            delete[] hostBacktrackMatrix;
        }

        hipFree(deviceSequences);
    #else
        char *referenceString = &sequences[sequenceIdxs[0].referenceIdx];
        char *queryString = &sequences[sequenceIdxs[0].queryIdx];
        // char *referenceString = "GTCATGCAATAACG";
        // char *queryString = "ATGCAATA";
        // char *referenceString = "GTCAGTA";
        // char *queryString = "ATACA";

        int referenceLength = strlen(referenceString);
        int queryLength = strlen(queryString);

        printf("Reference String: %s (Length: %d)\n", referenceString, referenceLength);
        printf("Query String: %s (Length: %d)\n", queryString, queryLength);
        printf("(MATCH WEIGHT, MISMATCH WEIGHT, GAP WEIGHT): (%d, %d, %d)\n\n", matchWeight, mismatchWeight, gapWeight);

        // Allocate device memory for matrices
        printf("[Allocating CUDA Memory]\n");
        int *deviceScoringMatrix;
        directionMain *deviceBacktrackMatrix;
        char *deviceReferenceString;
        char *deviceQueryString;

        handleErrs(
            hipMalloc(&deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int)),
            "FAILED TO ALLOCATE MEMORY TO SCORING MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACK MATRIX\n"
        );

        handleErrs(
            hipMalloc(&deviceReferenceString, (referenceLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceReferenceString, referenceString, (referenceLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO REFERENCE STRING\n"
        );

        handleErrs(
            hipMalloc(&deviceQueryString, (queryLength) * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO QUERY STRING\n"
        );

        handleErrs(
            hipMemcpy(deviceQueryString, queryString, (queryLength) * sizeof(char), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY TO QUERY STRING\n"
        );

        // Need to launch sinular kernel
        // Launching a kernel with 1 block with threadCount threads to populate scoring matrix
        needleman_wunsch_kernel<<<1, BLOCK_SIZE>>>(
            deviceScoringMatrix, deviceBacktrackMatrix,
            deviceQueryString, deviceReferenceString, 
            queryLength, referenceLength, 
            matchWeight, mismatchWeight, gapWeight
        );

        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );

        // Allocate host memory for matrices
        // Allow for matrices to come from device -> host
        // Free up device memory
        int *hostScoringMatrix = new int[(referenceLength+1) * (queryLength+1)];
        directionMain *hostBacktrackMatrix = new directionMain[(referenceLength+1) * (queryLength+1)];

        // Copy information back from device --> host
        handleErrs(
            hipMemcpy(hostScoringMatrix, deviceScoringMatrix, (referenceLength+1) * (queryLength+1) * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SCORING MATRIX FROM DEVICE --> HOST"
        );
        
        handleErrs(
            hipMemcpy(hostBacktrackMatrix, deviceBacktrackMatrix, (referenceLength+1) * (queryLength+1) * sizeof(directionMain), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACK MATRIX FROM DEVICE --> HOST"
        );

        hipFree(deviceScoringMatrix);
        hipFree(deviceBacktrackMatrix);
        hipFree(deviceQueryString);
        hipFree(deviceReferenceString);

        // Print Matrix
        printf("Scored Matrix\n");
        printMatrix(hostScoringMatrix, referenceLength + 1, queryLength + 1);
        printf("Backtrack Matrix\n");
        printBacktrackMatrix(hostBacktrackMatrix, referenceLength + 1, queryLength + 1);
        

        // Perform backtracking on host and print results
        printf("0 | %d\n", hostScoringMatrix[(referenceLength + 1) * (queryLength + 1) - 1]);
        backtrackNW(hostBacktrackMatrix, referenceString, referenceLength, queryString, queryLength);
        
        // Free data arrays
        delete[] hostScoringMatrix;
        delete[] hostBacktrackMatrix;
    #endif

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(sequenceIdxs, sequences);
}