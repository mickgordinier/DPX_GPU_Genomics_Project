#include "hip/hip_runtime.h"
#include <stdio.h>  // For printf()
#include <cstring> // Determining length of string
#include "../c++/parseInput.h"
#include "../c++/backtrack.h"
#include "../c++/timing.h"

// Blocks are 1D with a size of the 32 threads (For 1 warp)
#define BLOCK_SIZE 32
#define BATCH_SIZE 1000


__global__ void 
needleman_wunsch_kernel(
    int *similarityScores,
    int *stringSpacing,
    directionMain *batchBacktrackMatrices,
    int *batchIndices,
    int *stringStartingIndices,
    char *backtrackStringsRet, 
    const char *allSequences, const seqPair *allSequenceInfo,
    const int matchWeight, const int mismatchWeight, const int gapWeight,
    const int startingSequenceIdx, const int stringLengthMax)
{

    const int tid = threadIdx.x;

    extern __shared__ int warpEdgeScore[]; 

    // We are launching multiple blocks, each of a warp of threads
    // Each block handles their own sequence alignment
    // We index into the array to obtain the strings and length
    
    int batchIndex = batchIndices[blockIdx.x];
    directionMain *backtrackMatrix = batchBacktrackMatrices + batchIndex;

    const int sequenceIdx = startingSequenceIdx + blockIdx.x;
    const seqPair sequenceInfo = allSequenceInfo[sequenceIdx];
    
    const char *queryString = allSequences + sequenceInfo.queryIdx;
    const char *referenceString = allSequences + sequenceInfo.referenceIdx;

    const int queryLength = sequenceInfo.querySize;
    const int referenceLength = sequenceInfo.referenceSize;

    // The matrices are of size (queryLength + 1) * (referenceLength + 1)
    const int numRows = sequenceInfo.querySize + 1;
    const int numCols = sequenceInfo.referenceSize + 1;

    /* --- (BEGIN) INITIALIZING THE SCORING MATRIX --- */

    // Used for when a thread has to iterate over more than one col/row
    int elementIdx;

    // Initialize the top row
    // Writing in DRAM burst for faster updating
    elementIdx = tid;
    while(elementIdx < numCols) {
        backtrackMatrix[elementIdx] = QUERY_INSERTION;
        elementIdx += BLOCK_SIZE;
    }

    // Initialize the left col
    // NOT Writing in DRAM burst (slower)
    elementIdx = tid;
    while(elementIdx < numRows) {
        backtrackMatrix[elementIdx*numCols] = QUERY_DELETION;
        elementIdx += BLOCK_SIZE;
    }

    if (tid == 0) {
        backtrackMatrix[0] = NONE_MAIN;
    }

    // Need to ensure that all threads in the block complete filling up all the edges
    // Do not need to do syncthreads across each loop iteration as there is no dependencies
    __syncthreads();

    /* --- (END) INITIALIZING THE SCORING MATRIX --- */

    /* --- (BEGIN) POPULATING THE SCORING MATRIX -- */

    /*
    relative cell indices
    [00][01]
    [10][11]
    */

    int leftDiag = gapWeight*tid;
    int left = gapWeight*(tid+1);
    int up = gapWeight*(tid+1); 

    // Going through all of the rows each thread has to do
    for (int stripeStart = 1; stripeStart < numRows; stripeStart+=BLOCK_SIZE){

        int row = stripeStart + tid;
        int largestScore;

        /* threads outside of bounds should abort */
        if (row >= numRows) return;

        leftDiag = gapWeight*(row - 1);
        left = gapWeight*(row);

        for (int col = 1; col < (numCols+numRows); ++col){
            
            int adj_col = col - tid;

            if (row == 1){
                leftDiag = gapWeight*(adj_col - 1);
                up = gapWeight*(adj_col);
            }

            /* for all but the first stripe, t0 must grab its diagonal and upper values from t31 */
            if (stripeStart > 1 && tid == 0 && adj_col < numCols){
                up = warpEdgeScore[adj_col];
                leftDiag = (adj_col == 1) ? gapWeight*(row - 1) : warpEdgeScore[adj_col - 1];
            }

            if (adj_col > 0 && adj_col < numCols){
                largestScore = 0;
                char queryChar = queryString[row-1];
                char referenceChar = referenceString[adj_col-1];
    
                directionMain cornerDirection = NONE_MAIN;
                bool pred;
                bool isMatch = (queryChar == referenceChar);
                cornerDirection = isMatch ? MATCH : MISMATCH;
    
                int matchMismatchScore = isMatch ? leftDiag + matchWeight : leftDiag + mismatchWeight;
                int queryDeletionScore = up + gapWeight;
                int queryInsertionScore = left + gapWeight;
    
                largestScore = __vibmax_s32(queryDeletionScore, matchMismatchScore, &pred);
                if (pred) cornerDirection = QUERY_DELETION;
                        
                largestScore = __vibmax_s32(queryInsertionScore, largestScore, &pred);
                if (pred) cornerDirection = QUERY_INSERTION;

                // scoringMatrix[row * numCols + adj_col] = largestScore;
                backtrackMatrix[row * numCols + adj_col] = cornerDirection;

                left = largestScore;

                /* last thread in warp stores its scores in shared memory for t0 to access */
                if (tid == 31){
                    warpEdgeScore[adj_col] = largestScore;
                }

                leftDiag = up;
            }

            /*  top value for thread n + 1 is thread n's largestScore (just calculated value)*/
            up = __shfl_up_sync(0xffffffff, largestScore, 1);
        }

        if (row == numRows-1) {
            similarityScores[blockIdx.x] = largestScore;
        }
    }

    /* --- (END) POPULATING THE SCORING MATRIX -- */

    /* --- (BEGIN) DETERMINING BACKTRACKING -- */

    // Starting at the end
    if (tid == 0) {

        int jumpToNextString = queryLength + referenceLength + 1;

        int referenceStrIdx = stringStartingIndices[blockIdx.x] + (jumpToNextString - 1);
        int alignmentStrIdx = referenceStrIdx + jumpToNextString;
        int queryStrIdx = alignmentStrIdx + jumpToNextString;

        backtrackStringsRet[referenceStrIdx] = '\0';
        backtrackStringsRet[alignmentStrIdx] = '\0';
        backtrackStringsRet[queryStrIdx] = '\0';

        int currentMemoRow = numRows - 1;
        int currentMemoCol = numCols - 1;

        while ((currentMemoRow != 0) || (currentMemoCol != 0)) {

            referenceStrIdx--;
            alignmentStrIdx--;
            queryStrIdx--;
            
            // Determine the current cell's predecessor
            switch (backtrackMatrix[(currentMemoRow * numCols) + currentMemoCol]) {
                
                case MATCH:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '*';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if match

                case MISMATCH: 
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = '|';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    --currentMemoCol;
                    break;
                // end if mismatch
                
                case QUERY_DELETION:
                    backtrackStringsRet[referenceStrIdx] = '_';
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = queryString[currentMemoRow-1];
                    --currentMemoRow;
                    break;
                // end if query deletion
                
                case QUERY_INSERTION:
                    backtrackStringsRet[referenceStrIdx] = referenceString[currentMemoCol-1];
                    backtrackStringsRet[alignmentStrIdx] = ' ';
                    backtrackStringsRet[queryStrIdx] = '_';
                    --currentMemoCol;
                    break;
                // end if query insertion
                
                default:
                    printf("ERROR\n");
                    return;
                // end if upper gap

            } // end switch
        } // end while

        stringSpacing[blockIdx.x] = referenceStrIdx;
    }

}


void
handleErrs(
    hipError_t err,
    const char *errMsg) 
{
    if (err != hipSuccess) {
        printf(errMsg);
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    }
}


int main(int argc, char *argv[]) {

    // Print some cuda details
    printf("[Cuda Details]\n");
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("FAILED TO GET DEVICE COUNT\n");
        printf("CUDA test kernel error: %s\n", hipGetErrorString(err));
        exit(1);
    }

    printf("Device count: %d\n", deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
    printf("Concurrent kernels?: %d\n\n", deviceProp.concurrentKernels);

    // Check that YOU use it correctly
    if (argc < 2) {
		fprintf(stderr, "usage: main -pairs <InSeqFile> -match <matchWeight> -mismatch <mismatchWeight> -gap <gapWeight> \n");
		exit(EXIT_FAILURE);
    }
	
    // Get args
    char *pairFileName;
    int matchWeight     = 3;
    int mismatchWeight  = -1;
    int gapWeight       = -2;
    if(strcmp(argv[1], "-pairs") == 0) {
        pairFileName = argv[2];
    }
    if(argc > 3 && strcmp(argv[3], "-match") == 0) {
        matchWeight = atoi(argv[4]);
    }
    if(argc > 5 && strcmp(argv[5], "-mismatch") == 0) {
        mismatchWeight = atoi(argv[6]);
    }
    if(argc > 7 && strcmp(argv[7], "-gap") == 0) {
        gapWeight = atoi(argv[8]);
    }

    // Parse input file
    printf("Parsing input file: %s\n", pairFileName);
    inputInfo fileInfo;
    seqPair* allSequenceInfo;
    char* sequences;
    fileInfo = parseInput(pairFileName, allSequenceInfo, sequences);
    printf("Num Pairs: %d\n\n", fileInfo.numPairs);

    // Start timer
    uint64_t kernel_time = 0;
    uint64_t memalloc_time = 0;
    uint64_t backtracking_time = 0;
    uint64_t start_time = start_timer();
        
    // Copy over the sequences
    char* deviceSequences;
    seqPair *deviceAllSequenceInfo;

    uint64_t start_memalloc = get_time();
    handleErrs(
        hipMalloc(&deviceSequences, (fileInfo.numBytes) * sizeof(char)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceSequences, sequences, (fileInfo.numBytes) * sizeof(char), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMalloc(&deviceAllSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair)),
        "FAILED TO ALLOCATE MEMORY FOR ALL SEQUENCES\n"
    );

    handleErrs(
        hipMemcpy(deviceAllSequenceInfo, allSequenceInfo, (fileInfo.numPairs) * sizeof(seqPair), hipMemcpyHostToDevice),
        "FAILED TO COPY MEMORY FOR ALL SEQUENCES\n"
    );

    /* 
    store all backtracking matrices for a batch in one matrix - each warp will index in via index array
    */
    int *deviceBacktrackingIndices;
    int *hostBacktrackingIndices = (int *)malloc(BATCH_SIZE * sizeof(int));
    
    handleErrs(
        hipMalloc(&deviceBacktrackingIndices, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceBacktrackingIndices\n"
    );

    int *deviceStringStartingIndices;
    int *hostStringStartingIndices = (int *)malloc(BATCH_SIZE * sizeof(int));
    
    handleErrs(
        hipMalloc(&deviceStringStartingIndices, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO hostStringStartingIndices\n"
    );
    
    int *deviceSimilarityScores;
    int *hostSimilarityScores = (int*)malloc(BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceSimilarityScores, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );
    
    int *deviceStringSpacing;
    int *hostStringSpacing = (int*)malloc(BATCH_SIZE * sizeof(int));

    handleErrs(
        hipMalloc(&deviceStringSpacing, BATCH_SIZE * sizeof(int)),
        "FAILED TO ALLOCATE MEMORY TO deviceSimilarityScores\n"
    );

    memalloc_time += get_time() - start_memalloc;

    // Run the kernel on every sequence
    for(size_t sequenceIdx = 0; sequenceIdx < fileInfo.numPairs; sequenceIdx+=BATCH_SIZE){
        start_memalloc = get_time();

        int largestReferenceLength = 0;
        int largestQueryLength = 0;

        /* first warp's starting index is 0 */
        uint64_t batchMatrixSize = 0;
        uint64_t totalStringsSize = 0;
        hostBacktrackingIndices[0] = 0;
        hostStringStartingIndices[0] = 0;
        for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
            const int queryLength = allSequenceInfo[i].querySize;
            const int referenceLength = allSequenceInfo[i].referenceSize;

            largestReferenceLength = max(largestReferenceLength, referenceLength);
            largestQueryLength = max(largestQueryLength, queryLength);

            /* make sure we don't go over the end of the array */
            batchMatrixSize += ((referenceLength + 1) * (queryLength + 1));
            totalStringsSize += (3 * (queryLength + referenceLength + 1));

            if ((i - sequenceIdx) < (BATCH_SIZE - 1)){
                hostBacktrackingIndices[i-sequenceIdx + 1] = batchMatrixSize;
                hostStringStartingIndices[i-sequenceIdx + 1] = totalStringsSize;
            }
        }

        /* copy backtracking indices to device */
        handleErrs(
            hipMemcpy(deviceBacktrackingIndices, hostBacktrackingIndices, BATCH_SIZE * sizeof(int), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR deviceBacktrackingIndices\n"
        );

        handleErrs(
            hipMemcpy(deviceStringStartingIndices, hostStringStartingIndices, BATCH_SIZE * sizeof(int), hipMemcpyHostToDevice),
            "FAILED TO COPY MEMORY FOR deviceStringStartingIndices\n"
        );

        /* allocate device mem for all backtracking matrices */
        directionMain *deviceMatricesAll;
        handleErrs(
            hipMalloc(&deviceMatricesAll, batchMatrixSize*sizeof(directionMain)),
            "FAILED TO ALLOCATE MEMORY TO deviceMatricesAll\n"
        );

        char *deviceBacktrackStringRet;
        handleErrs(
            hipMalloc(&deviceBacktrackStringRet, totalStringsSize * sizeof(char)),
            "FAILED TO ALLOCATE MEMORY TO BACKTRACKING STRINGS\n"
        );

        int stringLengthMax = (largestReferenceLength+largestQueryLength+1);

        memalloc_time += get_time() - start_memalloc;

        uint64_t start_kernel = get_time();
        // Need to launch kernel
        int smem_size = (largestReferenceLength + 1) * sizeof(int);
        needleman_wunsch_kernel<<<BATCH_SIZE, BLOCK_SIZE, smem_size>>>(
            deviceSimilarityScores,
            deviceStringSpacing,
            deviceMatricesAll,
            deviceBacktrackingIndices,
            deviceStringStartingIndices,
            deviceBacktrackStringRet,
            deviceSequences, deviceAllSequenceInfo,
            matchWeight, mismatchWeight, gapWeight,
            sequenceIdx, stringLengthMax
        );
        
        // Wait for kernel to finish
        handleErrs(
            hipDeviceSynchronize(),
            "SYNCHRONIZATION FAILED\n"
        );
        kernel_time += get_time() - start_kernel;

        start_memalloc = get_time();

        handleErrs(
            hipMemcpy(hostSimilarityScores, deviceSimilarityScores, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
        );

        handleErrs(
            hipMemcpy(hostStringSpacing, deviceStringSpacing, BATCH_SIZE * sizeof(int), hipMemcpyDeviceToHost),
            "FAILED TO COPY SIMILARITY SCORES FROM DEVICE --> HOST\n"
        );

        char *hostBacktrackingStringRet = (char *)malloc(totalStringsSize * sizeof(char));

        handleErrs(
            hipMemcpy(hostBacktrackingStringRet, deviceBacktrackStringRet, totalStringsSize * sizeof(char), hipMemcpyDeviceToHost),
            "FAILED TO COPY BACKTRACKING STRING FROM DEVICE --> HOST\n"
        );

        memalloc_time += get_time() - start_memalloc;

        for (int i = sequenceIdx; i < sequenceIdx+BATCH_SIZE; ++i) {
        
            // Backtrack matrices
            printf("%d | %d\n", i, hostSimilarityScores[i-sequenceIdx]);

            const int queryLength = allSequenceInfo[i].querySize;
            const int referenceLength = allSequenceInfo[i].referenceSize;

            const int jumpToNextString = queryLength + referenceLength + 1;

            const int spacing = hostStringSpacing[i-sequenceIdx];

            printf("%s\n", hostBacktrackingStringRet + spacing);
            printf("%s\n", hostBacktrackingStringRet + jumpToNextString + spacing);
            printf("%s\n", hostBacktrackingStringRet + jumpToNextString + jumpToNextString + spacing);
        }

        free(hostBacktrackingStringRet);
        hipFree(deviceBacktrackStringRet);
        hipFree(deviceMatricesAll);
    }

    hipFree(deviceSequences);
    hipFree(deviceAllSequenceInfo);

    free(hostBacktrackingIndices);
    free(hostStringStartingIndices);
    free(hostSimilarityScores);
    free(hostStringSpacing);

    hipFree(deviceBacktrackingIndices);
    hipFree(deviceStringStartingIndices);
    hipFree(deviceSimilarityScores);
    hipFree(deviceStringSpacing);


    uint64_t elapsed_time = get_elapsed_time();
    printf("Elapsed time (usec): %lld\n", elapsed_time);
    printf("Elapsed kernel time (usec): %lld\n", kernel_time);
    printf("Elapsed backtracking time (usec): %lld\n", backtracking_time);
    printf("Elapsed memallocing time (usec): %lld\n", memalloc_time);
    printf("Elapsed time sum (usec): %lld\n",kernel_time + backtracking_time + memalloc_time);

    // Cleanup
    printf("Cleaning up\n");
    cleanupParsedFile(allSequenceInfo, sequences);
}